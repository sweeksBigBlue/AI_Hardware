
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <fstream>
#include <chrono>

__global__ void saxpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        y[i] = a * x[i] + y[i];
}

int main(void) {
    std::ofstream csv("saxpy_profiled_log.csv");
    csv << "log2_N,N,KernelTime_ms,TotalTime_ms,MaxError\n";

    for (int exp = 16; exp <= 29; exp++) {
        auto total_start = std::chrono::high_resolution_clock::now();

        unsigned long long N = 1ULL << exp;
        float *x = nullptr, *y = nullptr, *d_x = nullptr, *d_y = nullptr;

        x = (float *)malloc(N * sizeof(float));
        y = (float *)malloc(N * sizeof(float));
        if (!x || !y) {
            printf("Skipping N = 2^%d: malloc failed\n", exp);
            continue;
        }

        if (hipMalloc(&d_x, N * sizeof(float)) != hipSuccess ||
            hipMalloc(&d_y, N * sizeof(float)) != hipSuccess) {
            printf("Skipping N = 2^%d: hipMalloc failed\n", exp);
            free(x); free(y);
            continue;
        }

        for (unsigned long long i = 0; i < N; i++) {
            x[i] = 1.0f;
            y[i] = 2.0f;
        }

        hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

        // --- Measure kernel time only ---
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        int threadsPerBlock = 256;
        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
        saxpy<<<blocksPerGrid, threadsPerBlock>>>(N, 2.0f, d_x, d_y);

        hipError_t errSync = hipDeviceSynchronize();
        hipError_t errAsync = hipGetLastError();
        if (errSync != hipSuccess) printf("Sync error at N=2^%d: %s\n", exp, hipGetErrorString(errSync));
        if (errAsync != hipSuccess) printf("Async error at N=2^%d: %s\n", exp, hipGetErrorString(errAsync));

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float kernel_ms = 0.0f;
        hipEventElapsedTime(&kernel_ms, start, stop);

        hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

        float maxError = 0.0f;
        for (unsigned long long i = 0; i < N; i++) {
            maxError = fmaxf(maxError, fabsf(y[i] - 4.0f));
        }

        auto total_end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> total_ms = total_end - total_start;

        printf("N = 2^%d (%llu): Kernel = %.3f ms, Total = %.3f ms, Max error = %f\n",
               exp, N, kernel_ms, total_ms.count(), maxError);

        csv << exp << "," << N << "," << kernel_ms << "," << total_ms.count() << "," << maxError << "\n";

        // Cleanup
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(d_x);
        hipFree(d_y);
        free(x);
        free(y);
    }

    csv.close();
    return 0;
}
