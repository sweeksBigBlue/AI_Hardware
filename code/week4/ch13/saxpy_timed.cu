
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>  // for fabsf()

__global__ void saxpy(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        y[i] = a * x[i] + y[i];
}

int main(void) {
    // Sweep N from 2^16 to 2^25
    for (int exp = 16; exp <= 25; exp++) {
        int N = 1 << exp;
        float *x, *y, *d_x, *d_y;
        
        // Allocate host memory
        x = (float *)malloc(N * sizeof(float));
        y = (float *)malloc(N * sizeof(float));
        
        // Allocate device memory
        hipMalloc(&d_x, N * sizeof(float));
        hipMalloc(&d_y, N * sizeof(float));
        
        // Initialize x and y arrays
        for (int i = 0; i < N; i++) {
            x[i] = 1.0f;
            y[i] = 2.0f;
        }
        
        hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

        // Create CUDA events for timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        // Record start event
        hipEventRecord(start);

        // Launch kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
        saxpy<<<blocksPerGrid, threadsPerBlock>>>(N, 2.0f, d_x, d_y);

        // Record stop event
        hipEventRecord(stop);

        // Wait for the event to complete
        hipEventSynchronize(stop);

        // Calculate elapsed time
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        // Copy result back to host
        hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

        // Compute maximum error
        float maxError = 0.0f;
        for (int i = 0; i < N; i++)
            maxError = fmaxf(maxError, fabsf(y[i] - 4.0f));

        // Output results
        printf("N = 2^%d (%d elements): Execution time = %.3f ms, Max error = %f\n",
               exp, N, milliseconds, maxError);

        // Clean up
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(d_x);
        hipFree(d_y);
        free(x);
        free(y);
    }

    return 0;
}
